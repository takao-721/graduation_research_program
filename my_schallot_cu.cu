#include "hip/hip_runtime.h"
/*
 CUDA版eschalot
卒業研究用
eschalot
*/

#include <stdio.h>
#include <string.h>
#include <stdint.h>

#include <time.h>

#include"hip/hip_runtime.h"

//OpenSSL---BN_new();,BN_set_word();
#include <openssl/bn.h>
//OpenSSL---RSA_generate_key();
#include <openssl/rsa.h>
//OpenSSL---i2d_RSAPublicKey();
#include <openssl/rsa.h>
#include <openssl/x509.h>
#include <openssl/engine.h>

#define __THREADS_X 128
#define __THREADS_Y 1
#define __THREADS_Z 1
#define __BLOCKS_Y 1

#define BN_PUT(bn) { printf(#bn "=%s (0x%s)\n", BN_bn2dec(bn), BN_bn2hex(bn)); }

#define BASE32_ALPHABET	"abcdefghijklmnopqrstuvwxyz234567"

//#define SHA1CircularShift(n,x) (((x)<<(n))|((x)>>(32-(n))))

__global__ void __calc_kernel(uint8_t *a1, int *b1,char *tank);

int main(){
  clock_t start,end;
  int counter=0;
  start = clock();
  uint8_t *tmp;
  signed int derlen;
  RSA *rsa = NULL;
  int loop=0;
  int cou_f=0;
  int n=10;

  //ファイル読み込みと比較用変数
  FILE *fp;
  int i=0,z=0;
  char ch;//tank[37122][17];
  //char *p;
  char *__host_tank;
  char *__dev_tank;

  BIGNUM *rsa_d;
  BIGNUM *rsa_e;

  uint8_t *__host_a1;
  uint8_t *__dev_a1;

  int *__host_b1;
  int *__dev_b1;

  __host_a1 = (uint8_t *) malloc(sizeof(uint8_t) * n * 16 + 1);
  __host_b1 = (int *) malloc(sizeof(int) * 1024);

  rsa_d=(BIGNUM *) malloc(sizeof(BIGNUM) * n);
  rsa_e=(BIGNUM *) malloc(sizeof(BIGNUM) * n);
  //printf("%d\n",sizeof(BIGNUM));
  //printf("test64\n");
  __host_tank = (char *)malloc(sizeof(char) * 37122 * 16 + 1);

  //printf("test67\n");
  if((fp=fopen("190413.txt","r"))==NULL){
    printf("Not open file!\n");
  }else{
    while((ch = fgetc(fp)) != EOF){
      __host_tank[i]=ch;
      if(__host_tank[i]=='.'){
        __host_tank[i]=' ';
        i--;
        //.onion\n分飛ばす
        while(z!=6){
          ch = fgetc(fp);
          z++;
        }
        z=0;
      }
      i++;
      if(i==37122*16+1) break;
    }
  }

  while(loop!=1){
    //    printf("test90\n");
    for(cou_f=0;cou_f<n;cou_f++){
      //RSAの公開鍵生成--------------------------------
      //RSA_KEYS_BITLEN---1024  RSA_E_START---0xFFFFFFu + 2
      rsa = RSA_generate_key(1024, 0xFFFFFFu + 2,NULL, NULL);
      rsa_d[cou_f]=*(rsa->d);
      rsa_e[cou_f]=*(rsa->e);

      //DERエンコード----------------------------------
      if((derlen = i2d_RSAPublicKey(rsa, NULL)) < 0)
			   printf("DER encoding failed!\n");
		  if ((tmp = (uint8_t *)malloc(derlen)) == NULL)
			   printf("malloc(derlen) failed!\n");
		  if (i2d_RSAPublicKey(rsa, &tmp) != derlen)
			   printf("DER encoding failed!\n");
      //printf("通っていますよ\n");
      __host_a1[cou_f]=*tmp;
    }


      //printf("test110\n");
    //n 個のグリッド生成
    dim3 __block(__THREADS_X, __THREADS_Y, __THREADS_Z);
    dim3 __grid(8, __BLOCKS_Y,__BLOCKS_Y);
    //printf("test114\n");

    //GPU 側のメモリ確保
    hipMalloc((void **) &__dev_a1, (sizeof(uint8_t) * n * 16 + 1));
    //printf("%d\n",&__dev_a1);
    hipMalloc((void **) &__dev_b1, (sizeof(int) * 1024));
    //printf("%d\n",&__dev_b1);
    hipMalloc((void **) &__dev_tank, (sizeof(char) * n *37122 * 16 + 1));
    //printf("%d\n",&__dev_tank);printf("test120\n");

    //データ転送
    hipMemcpy(__dev_a1, __host_a1, (sizeof(uint8_t) * n * 16 + 1),hipMemcpyHostToDevice);
    //printf("%d\n",&__dev_a1);
    hipMemcpy(__dev_b1, __host_b1, (sizeof(int) * 1024),hipMemcpyHostToDevice);
    //printf("%d\n",&__dev_b1);
    hipMemcpy(__dev_tank, __host_tank, (sizeof(char) * n * 37122 * 16 * 2 + 1),hipMemcpyHostToDevice);
    //printf("%d\n",&__dev_tank);
    //printf("test 127\n");

    //カーネル関数呼出し
    __calc_kernel <<< __grid, __block >>> (__dev_a1, __dev_b1, __dev_tank);
    //printf("test131\n");
    //CPU 側に値を返してくる
    hipMemcpy(__host_a1, __dev_a1, (sizeof(uint8_t) * n * 16 + 1),hipMemcpyDeviceToHost);
    hipMemcpy(__host_b1, __dev_b1, (sizeof(int) * 1024),hipMemcpyDeviceToHost);

    for(i=0;i<n;i++){
      if(__host_b1[i]==1){
         printf("秘密鍵d\n");
         // BN_PUT(*(rsa_d+i));
         // printf("秘密鍵e\n");
         // BN_PUT(*(rsa_e+i));
         return 0;
      }
    }

    counter=counter+i;
    //printf("test136\n");
    if(counter>=100000){
      end=clock();
      printf("%.2f秒かかりました\n",(double)(end-start)/CLOCKS_PER_SEC );
    }

   }
   free(__host_a1);
   hipFree(__dev_a1);
   free(__host_b1);
   hipFree(__dev_b1);
   free(__host_tank);
   hipFree(__dev_tank);

   return 0;
}


__global__ void __calc_kernel(uint8_t *a1, int *b1,char *tank){
  // unsigned int __tmp_idx_x = (threadIdx.x + (blockDim.x * blockIdx.x));
  // unsigned int __tmp_idx_y = (threadIdx.y + (blockDim.y * blockIdx.y));
  // unsigned int __tmp_size_x = (blockDim.x * gridDim.x);
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  // unsigned int id = (__tmp_idx_x + (__tmp_idx_y * __tmp_size_x));
  //SHA1ハッシュ-----------------------------------
  //SHA1変数----------------------------------
  uint8_t buf[20],onion[17];
  uint32_t W[80]={0};
  uint32_t a,b,c,d,e;
  uint32_t H[5]={0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476, 0xC3D2E1F0};
  const uint32_t K[4]={0x5A827999,0x6ED9EBA1,0x8F1BBCDC,0xCA62C1D6};
  uint32_t temp;
  uint32_t result_H[5];

  int i=0,j=0,hyouzi=0;
  char con[16];

  //printf("test\n");

 //printf("通っていますね\n");

 //ポインタの配列とただの配列[i-1]は同じ意味か？
 //uint8_t plain[] ==uint8_t *tmp
 // tmp[i-1]
  j=0;  //これがないと値が同じになってしまう
  W[64]=0;  //これがないと値が同じになってしまう
  for(i=1;i<=16;i++){
    W[j]|=a1[i-1]<<8*((j+1)*4-i);
    if(i%4==0){
      j++;
    }
  }

  W[16/4] |= 0x8<<(4+(7-(16%4)*2));
  W[15]|=16*8;

  W[64]=0;

  // n,x --- (n,x) --- (((x)<<(n))|((x)>>(32-(n))))

  for(i=16;i<80;i++){
    W[i]=(((W[i-3]^W[i-8]^W[i-14]^W[i-16])<<(1))|((W[i-3]^W[i-8]^W[i-14]^W[i-16])>>(32-(1))));//SHA1CircularShift(1,W[i-3]^W[i-8]^W[i-14]^W[i-16]);
  }

  a=H[0];
  b=H[1];
  c=H[2];
  d=H[3];
  e=H[4];

  // c=が抜けると正しく計算できない

  //&---and演算 ~---各ビットの反転
  for(j=0;j<20;j++){
    temp=(((a)<<(5))|((a)>>(32-(5))))+((b&c)|((~b)&d))+e+K[0]+W[j];//SHA1CircularShift(5,a)+((b&c)|((~b)&d))+e+K[0]+W[j];
    e=d;
    d=c;
    c=(((b)<<(30))|((b)>>(32-(30))));//SHA1CircularShift(30,b);
    b=a;
    a=temp;
  }

  for(j=20;j<40;j++){
    temp=(((a)<<(5))|((a)>>(32-(5))))+(b^c^d)+e+K[1]+W[j];//SHA1CircularShift(5,a)+(b^c^d)+e+K[1]+W[j];
    e=d;
    d=c;
    c=(((b)<<(30))|((b)>>(32-(30))));//SHA1CircularShift(30,b);
    b=a;
    a=temp;
  }

  for(j=40;j<60;j++){
    temp=(((a)<<(5))|((a)>>(32-(5))))+((b&c)|(b&d)|(c&d))+e+K[2]+W[j];//SHA1CircularShift(5,a)+((b&c)|(b&d)|(c&d))+e+K[2]+W[j];
    e=d;
    d=c;
    c=(((b)<<(30))|((b)>>(32-(30))));//SHA1CircularShift(30,b);
    b=a;
    a=temp;
  }

  for(j=60;j<80;j++){
    temp=(((a)<<(5))|((a)>>(32-(5))))+(b^c^d)+e+K[3]+W[j];//SHA1CircularShift(5,a)+(b^c^d)+e+K[3]+W[j];
    e=d;
    d=c;
    c=(((b)<<(30))|((b)>>(32-(30))));//SHA1CircularShift(30,b);
    b=a;
    a=temp;
  }
  //result_H---32bit
  result_H[0]=a+H[0];
  result_H[1]=b+H[1];
  result_H[2]=c+H[2];
  result_H[3]=d+H[3];
  result_H[4]=e+H[4];

 //ビッグエンディアン
  buf[0] = result_H[0]>>24;
  buf[1] = result_H[0]>>16;
  buf[2] = result_H[0]>>8;
  buf[3] = result_H[0];
  buf[4] = result_H[1]>>24;
  buf[5] = result_H[1]>>16;
  buf[6] = result_H[1]>>8;
  buf[7] = result_H[1];
  buf[8] = result_H[2]>>24;
  buf[9] = result_H[2]>>16;

  // Base32
  onion[ 0] = BASE32_ALPHABET[ (buf[0] >> 3)	];
  onion[ 1] = BASE32_ALPHABET[((buf[0] << 2) | (buf[1] >> 6))	& 31];
  onion[ 2] = BASE32_ALPHABET[ (buf[1] >> 1) & 31];
  onion[ 3] = BASE32_ALPHABET[((buf[1] << 4) | (buf[2] >> 4))	& 31];
  onion[ 4] = BASE32_ALPHABET[((buf[2] << 1) | (buf[3] >> 7))	& 31];
  onion[ 5] = BASE32_ALPHABET[ (buf[3] >> 2)	& 31];
  onion[ 6] = BASE32_ALPHABET[((buf[3] << 3) | (buf[4] >> 5))	& 31];
  onion[ 7] = BASE32_ALPHABET[  buf[4]& 31];

  onion[ 8] = BASE32_ALPHABET[ (buf[5] >> 3)	];
  onion[ 9] = BASE32_ALPHABET[((buf[5] << 2) | (buf[6] >> 6))	& 31];
  onion[10] = BASE32_ALPHABET[ (buf[6] >> 1)& 31];
  onion[11] = BASE32_ALPHABET[((buf[6] << 4) | (buf[7] >> 4))	& 31];
  onion[12] = BASE32_ALPHABET[((buf[7] << 1) | (buf[8] >> 7))	& 31];
  onion[13] = BASE32_ALPHABET[ (buf[8] >> 2)& 31];
  onion[14] = BASE32_ALPHABET[((buf[8] << 3) | (buf[9] >> 5))	& 31];
  onion[15] = BASE32_ALPHABET[  buf[9]& 31];

  onion[16] = '\0';

  //printf("通っています2\n");

  // for(i=0;i<16;i++){
  //   printf("%c\n",onion[i]);
  // }
  // printf("\n");

  for(i=0;i<16;i++){
    con[i]=(char)onion[i];
  }

    //printf("test296\n");
  //比較
  for(i=0;i<37122*16+1;i++){
    if(*(tank+i)==con[j]){
      j++;
      if(j==15){
        printf("発見\n");
        j=j-15;
        while(hyouzi<16){
          printf("%c\n",con[j]);
          b1[idx]=1;
          hyouzi++;
          j++;
        }
        printf("\n");
        j=0;
      }
    }else{
     j=0;
    }
  }
  //printf("test");
}
